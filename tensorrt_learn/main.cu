#include <string>
#include <vector>

#include <NvInfer.h>
#include <NvInferPlugin.h>
#include <NvInferVersion.h>
#include <hip/hip_runtime.h>

#include "tensorrt_utils.cuh"

int main(int argc, char const* argv[]) {
    auto model = Ahri::TensorRT::Model("/media/supervisor/windowsd/Code/github/Vision/tensorrt_learn/ahrinet.onnx");
    model.build();
    model.infer();

    return 0;
}
